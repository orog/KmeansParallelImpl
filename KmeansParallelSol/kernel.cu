#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void distanceWithCuda(double* points, int numOfPoints, double* clustersCenters, int numOfClusters, int dimension, double* results, int pointsPerBlock)
{
	int i, blockID = blockIdx.x;
	double result = 0;

	if (blockID == gridDim.x - 1 && numOfPoints % blockDim.x <= threadIdx.x)
		return;

	for (i = 0; i < dimension; i++)
	{
		result += (points[(blockID*pointsPerBlock + threadIdx.x)*dimension + i] - clustersCenters[threadIdx.y*dimension + i]) *  (points[(blockID*pointsPerBlock + threadIdx.x)*dimension + i] - clustersCenters[threadIdx.y*dimension + i]);
	}
	results[numOfPoints*threadIdx.y + (blockID*pointsPerBlock + threadIdx.x)] = result;
}

__global__ void findClosestCluster(double* distances, int numOfClusters, int numOfPoints, int pointsPerBlock, int* belongTo)
{
	int i, xid = threadIdx.x, blockId = blockIdx.x;
	double minIndex = 0, minDistance, tempDistance;

	if (blockIdx.x == gridDim.x - 1 && numOfPoints % blockDim.x <= xid)
		return;

	minDistance = distances[pointsPerBlock*blockId + xid];

	for (i = 1; i < numOfClusters; i++)
	{
		tempDistance = distances[pointsPerBlock*blockId + xid + i*numOfPoints];
		if (minDistance > tempDistance)
		{
			minIndex = i;
			minDistance = tempDistance;
		}
	}
	belongTo[pointsPerBlock*blockId + xid] = minIndex;
}

hipError_t dividePointToClustersWithCuda(double* pointsCoordinationsPointer, int numOfPoints, double* clustersCentersCoordinations, int numOfClusters, int dimension, int** belongTo)
{
	double *dev_clustersCentersCoordinations = NULL;
	double *dev_distances = NULL;
	int pointsPerBlock, numBlocks, i, extraBlock , *dev_belongTo = NULL;
	hipDeviceProp_t prop;
	hipError_t cudaStatus;

	*belongTo = (int*)calloc(numOfPoints, sizeof(int));

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	handleErrors(cudaStatus, "hipSetDevice failed!");

	cudaStatus = hipGetDeviceProperties(&prop, 0);
	handleErrors(cudaStatus, "hipGetDeviceProperties failed!");

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_clustersCentersCoordinations, dimension * numOfClusters * sizeof(double));
	handleErrors(cudaStatus, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&dev_distances, numOfPoints * numOfClusters * sizeof(double));
	handleErrors(cudaStatus, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&dev_belongTo, numOfPoints * sizeof(int));
	handleErrors(cudaStatus, "hipMalloc failed!");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_clustersCentersCoordinations, clustersCentersCoordinations, dimension * numOfClusters * sizeof(double), hipMemcpyHostToDevice);
	handleErrors(cudaStatus, "hipMemcpy failed!");


	// Launch a kernel on the GPU with one thread for each element.
	pointsPerBlock = prop.maxThreadsPerBlock / numOfClusters;
	dim3 dim(pointsPerBlock, numOfClusters);
	numBlocks = numOfPoints / pointsPerBlock;
	if (numOfPoints % pointsPerBlock == 0)
		extraBlock = 0;
	else
		extraBlock = 1;
	distanceWithCuda << <numBlocks + extraBlock, dim >> > (pointsCoordinationsPointer, numOfPoints, dev_clustersCentersCoordinations, numOfClusters, dimension, dev_distances, pointsPerBlock);

	cudaStatus = hipDeviceSynchronize();
	handleErrors(cudaStatus, "cudaDeviceSynchronize1 failed!\n");

	pointsPerBlock = prop.maxThreadsPerBlock;
	numBlocks = numOfPoints / pointsPerBlock;
	if (numOfPoints % pointsPerBlock == 0)
		extraBlock = 0;
	else
		extraBlock = 1;
	findClosestCluster << <numBlocks + extraBlock, pointsPerBlock >> > (dev_distances, numOfClusters, numOfPoints, pointsPerBlock, dev_belongTo);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	handleErrors(cudaStatus, "addKernel launch failed: %s\n");


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	handleErrors(cudaStatus, "cudaDeviceSynchronize2 failed!\n");



	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(*belongTo, dev_belongTo, numOfPoints * sizeof(int), hipMemcpyDeviceToHost);
	handleErrors(cudaStatus, "hipMemcpy failed!");
	//hipFree(dev_pointsCordinations);
	hipFree(dev_clustersCentersCoordinations);
	hipFree(dev_distances);
	hipFree(dev_belongTo);
	return cudaStatus;
}

hipError_t copyPointsCordToCUDA(double* pointsCoordinations, int numOfPoints,int dimension , double** pointsCoordCUDAPointer)
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	handleErrors(cudaStatus, "copyPointsCord hipSetDevice failed!\n");

	cudaStatus = hipMalloc((void**)pointsCoordCUDAPointer, numOfPoints*dimension * sizeof(double));
	handleErrors(cudaStatus, "copyPointsCord hipMalloc failed!\n");
	cudaStatus = hipMemcpy(*pointsCoordCUDAPointer, pointsCoordinations, numOfPoints*dimension * sizeof(double), hipMemcpyHostToDevice);
	handleErrors(cudaStatus, "copyPointsCord hipMemcpy failed!\n");
	return cudaStatus;
}



void handleErrors(hipError_t cudaStatus, const char* errorMessage)
{
	if (cudaStatus != hipSuccess)
	{
		printf(errorMessage);
		fflush(stdout);
		system("pause");
		exit(1);
	}
}
